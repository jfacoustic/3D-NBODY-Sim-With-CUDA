#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <vector>
#include "ShaderClass.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>
#include <cstdlib>
#include "initializeScreen.hpp"
#include "glvec6.hpp"
#include "particles.hpp"
#include "spheres.hpp"
#include "camera.hpp"
//GLEW
#define GLEW_STATIC
#include <GL/glew.h>

//GLFW
#include <GLFW/glfw3.h>

#define n 1000
#define G 0.00000000000667408
#define dt 60*60
#define len 1000000.0
using namespace std;

GLfloat deltaTime = 0.0f;
GLfloat lastFrame = 0.0f;



GLuint screenWidth = 1.2 * 800;
GLuint screenHeight = 1.2 * 600;

int N = 0;




__device__ void move(particle &a, float3 accel) {
	a.x.x = a.x.x + a.v.x * dt;
	a.x.y = a.x.y + a.v.y * dt;
	a.x.z = a.x.z + a.v.z * dt;
	a.v.x = a.v.x + accel.x * dt;
	a.v.y = a.v.y + accel.y * dt;
	a.v.z = a.v.z + accel.z * dt;

}

__device__ float3 gravity(particle a, particle b) {
	float3 r = {b.x.x - a.x.x, b.x.y - a.x.y, b.x.z - a.x.z};
	float rabs = sqrt(r.x * r.x + r.y * r.y + r.z * r.z);
	float temp = G * b.m / (rabs * rabs * rabs);
	float3 accel = {temp*r.x, temp*r.y, temp*r.z};

	return accel;
}



void initializeParticles(particle *a) {
	for(int i = 0; i < n; i++) {
		bool works = false;
		while (!works) {
		a[i].m = 1.0f;
		a[i].x.x = -20.0f + static_cast <float> (rand()) / static_cast<float> (RAND_MAX /(40.0f));
		a[i].x.y = -20.0f + static_cast <float> (rand()) / static_cast<float> (RAND_MAX / 40.0f);
		a[i].x.z = -20.0f + static_cast <float> (rand()) / static_cast<float> (RAND_MAX / 40.0f);
		a[i].v.x = 0.0f;
		a[i].v.y = 0.0f;
		a[i].v.z = 0.0f;
		if (a[i].x.x * a[i].x.x + a[i].x.y * a[i].x.y + a[i].x.z * a[i].x.z < 20 * 20) works = true;
		}
	}
}

__global__ void updateParticles(particle *a) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n) {
		float3 accel = {0.0, 0.0, 0.0};
		for (int i = 0; i < n; i ++) {
			if (i != tid) {

				float3 temp = gravity(a[tid], a[i]);
				accel.x += temp.x;
				accel.y += temp.y;
				accel.z += temp.z;
			}
		}
		move(a[tid], accel);
		tid += gridDim.x;
	}
}

void key_callback(GLFWwindow * window, int key, int scancode, int action, int mode);
void mouse_callback(GLFWwindow* window, double xpos, double ypos);
void scroll_callback(GLFWwindow *window, double xoffset, double yoffset);
void do_movement();

Camera camera(glm::vec3(0.0f, 0.0f, 10.0f));
GLfloat lastX = screenWidth / 2.0;
GLfloat lastY = screenHeight / 2.0;
bool keys[1024];

int main() {
	GLFWwindow * window;
	initilizeScreen(window, screenWidth, screenHeight);
    
    
    
    glewExperimental = GL_TRUE; //Ensures Glew uses modern techniques for managing OGL
    if (glewInit() != GLEW_OK)
    {
        std::cout << "Failed to initialize GLEW" << std::endl;
        return -1;
    }
    
    GLint nrAttributes;
    glGetIntegerv(GL_MAX_VERTEX_ATTRIBS, &nrAttributes );
    std::cout << "Vertex Attributes: " << nrAttributes << std::endl;
    
    
    //register callback functions after window created but before game loop!
    glfwSetKeyCallback(window, key_callback);
    glfwSetCursorPosCallback(window, mouse_callback);
    glfwSetScrollCallback(window, scroll_callback);

    glViewport(0, 0, screenWidth, screenHeight); //location of lower left corner followed by width and height
    
    Shader ourShader("vertex.shader" , "fragment.shader");
    GLuint VAO, VBO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glBindVertexArray(VAO);
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    
    int prec = 3;

    
    createSphere(VAO, VBO, prec);
    
    Shader lightingShader("lightingVertex.shader", "lightingFragment.shader");
    Shader lampShader("lampVertex.shader", "lampFragment.shader");
    
    
    particle *a;
	a = (particle*)malloc(n*sizeof(particle) );
	/*
	a[0].m = 3.0f;
	a[1].m = 1.0f;
	a[0].x.x = 0;
	a[0].x.y = 0;
	a[0].x.z = 0;
	a[0].v.x = 0.0f;
	a[0].v.y = 0;
	a[0].v.z = 0;
	a[1].x.x = 2.5f;
	a[1].x.y = 0;
	a[1].x.z = 1.0f;
	a[1].v.x = 3.0f;
	a[1].v.y = 1.5f;
	a[1].v.z = 0;
	a[2].x.x = -5.0f;
	a[2].x.y = 0.0f;
	a[2].x.z = 0.0f;
	a[2].v.x = 0.0f;
	a[2].v.y = -1.5f;
	a[2].v.z = 0.0f;
	a[2].m = 1.0f;*/
	initializeParticles(a);
	
	
	for (int i = 0; i < n; i++) {
		float3 accel = {0.0f, 0.0f, 0.0f};
		for (int j = 0; j < n; j++) {
			if(i != j) {
				float rx = a[j].x.x - a[i].x.x;
				float ry = a[j].x.y - a[i].x.y;
				float rz = a[j].x.z - a[i].x.z;
				float r = sqrt(pow((rx),2) + pow(ry,2) + pow(rz,2));
				accel.x += G * a[j].m * rx / (r*r*r);
				
				accel.y += G * a[j].m * ry / (r*r*r);
				
				accel.z += G * a[j].m * rz / (r*r*r);
			}
			a[i].v.x += accel.x * dt / 2.0f;
			a[i].v.y += accel.y * dt / 2.0f;
			a[i].v.z += accel.z * dt / 2.0f;
		}
	}
	
	particle *a_dev;
	hipMalloc((void**)&a_dev, sizeof(particle) * n);
	hipMemcpy(a_dev, a, sizeof(particle) *n, hipMemcpyHostToDevice);
    //initializeParticles<<<128,128>>>(a);
	//hipMemcpy(a, a_dev, sizeof(particle) * n, hipMemcpyDeviceToHost);
    glEnable(GL_DEPTH_TEST);
    int l = 0;
    //game loop:
    while(!glfwWindowShouldClose(window))
    {
		GLfloat currentFrame = glfwGetTime();
		deltaTime = currentFrame - lastFrame;
		lastFrame = currentFrame;
		
       
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        //check and call events
        glfwPollEvents();
        do_movement();
        //rendering commands:
        glClearColor(0.1f, 0.1f, 0.1f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);
        updateParticles<<<128,128>>>(a_dev);
        //if(l == 1) {
        hipMemcpy(a, a_dev, sizeof(particle)*n,hipMemcpyDeviceToHost); l=0;//}
        l++;
        cout <<a[0].x.x << " " << a[1].x.x << endl;
		//set up shader
        lightingShader.Use();
        GLint objectColorLoc = glGetUniformLocation(lightingShader.Program, "objectColor");
        GLint lightColorLoc = glGetUniformLocation(lightingShader.Program, "lightColor");
        GLint lightPosLoc = glGetUniformLocation(lightingShader.Program, "lightPos");
        glUniform3f(objectColorLoc, 160.0f/255.0f, 82.0f/255.0f, 45.0f/255.0f);
        glUniform3f(lightColorLoc, 1.0f, 1.0f, 1.0f);
        glUniform3f(lightPosLoc, 0.0f, 0.0f, 100.0f);
        
        
        glBindVertexArray(VAO);
        
        GLfloat timeValue = 0 * glfwGetTime();
        
        
        
        glm::mat4 view;
        view = camera.GetViewMatrix();
        
		
        glm::mat4 projection;
        projection = glm::perspective(camera.Zoom, (float)screenWidth / (float)screenHeight, 0.1f, 1000.0f);
        
        GLint modelLoc = glGetUniformLocation(lightingShader.Program, "model");
        GLint projLoc = glGetUniformLocation(lightingShader.Program, "projection");
        GLint viewLoc = glGetUniformLocation(lightingShader.Program, "view");
        
        
        
        glUniformMatrix4fv(projLoc, 1, GL_FALSE, glm::value_ptr(projection));
        glUniformMatrix4fv(viewLoc, 1, GL_FALSE, glm::value_ptr(view));

        //GLint colorOffset = glGetUniformLocation(ourShader.Program, "colorOffset");
        //glUniform3f(colorOffset, std::abs(sin(0.5 * timeValue)), std::abs(cos(0.7*timeValue + 0.5)), std::abs(sin(timeValue)));
        
	    for (int i = 0; i < n; i++) {
            glm::mat4 model;
            model = glm::translate(model, glm::vec3(a[i].x.x, a[i].x.y, a[i].x.z));
            model = glm::scale(model, glm::vec3(0.25f, 0.25f, 0.25f));
            glUniformMatrix4fv(modelLoc, 1, GL_FALSE, glm::value_ptr(model));
            glDrawArrays(GL_TRIANGLES, 0,(GLsizei)triangles.size() * (GLsizei)sizeof(triangles));
        }
        
        
		glBindVertexArray(0);
		
        glfwSwapBuffers(window);
        
    }
    
    hipFree(a_dev);
	free(a);
    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);

    glfwTerminate();
    return 0;
}

void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode)
{
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, GL_TRUE);
    if (key >= 0 && key < 1024)
    {
        if (action == GLFW_PRESS)
            keys[key] = true;
        else if (action == GLFW_RELEASE)
            keys[key] = false;
    }
}

void do_movement()
{
    // Camera controls
    if (keys[GLFW_KEY_W])
        camera.ProcessKeyboard(FORWARD, deltaTime);
    if (keys[GLFW_KEY_S])
        camera.ProcessKeyboard(BACKWARD, deltaTime);
    if (keys[GLFW_KEY_A])
        camera.ProcessKeyboard(LEFT, deltaTime);
    if (keys[GLFW_KEY_D])
        camera.ProcessKeyboard(RIGHT, deltaTime);
}

bool firstMouse = true;
void mouse_callback(GLFWwindow* window, double xpos, double ypos)
{
    if (firstMouse)
    {
        lastX = xpos;
        lastY = ypos;
        firstMouse = false;
    }

    GLfloat xoffset = xpos - lastX;
    GLfloat yoffset = lastY - ypos;  // Reversed since y-coordinates go from bottom to left

    lastX = xpos;
    lastY = ypos;

    camera.ProcessMouseMovement(xoffset, yoffset);
}

void scroll_callback(GLFWwindow* window, double xoffset, double yoffset)
{
    camera.ProcessMouseScroll(yoffset);
}
	
